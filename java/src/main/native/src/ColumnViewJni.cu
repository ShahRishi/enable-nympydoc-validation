/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/labeling/label_segments.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/stream_compaction.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <rmm/exec_policy.hpp>
#include <thrust/scan.h>

#include "ColumnViewJni.hpp"

namespace cudf::jni {

std::unique_ptr<cudf::column>
new_column_with_boolean_column_as_validity(cudf::column_view const &exemplar,
                                           cudf::column_view const &validity_column) {
  CUDF_EXPECTS(validity_column.type().id() == type_id::BOOL8,
               "Validity column must be of type bool");
  CUDF_EXPECTS(validity_column.size() == exemplar.size(),
               "Exemplar and validity columns must have the same size");

  auto validity_device_view = cudf::column_device_view::create(validity_column);
  auto validity_begin = cudf::detail::make_optional_iterator<bool>(
      *validity_device_view, cudf::nullate::DYNAMIC{validity_column.has_nulls()});
  auto validity_end = validity_begin + validity_device_view->size();
  auto [null_mask, null_count] =
      cudf::detail::valid_if(validity_begin, validity_end, [] __device__(auto optional_bool) {
        return optional_bool.value_or(false);
      });
  auto const exemplar_without_null_mask = cudf::column_view{
      exemplar.type(),
      exemplar.size(),
      exemplar.head<void>(),
      nullptr,
      0,
      exemplar.offset(),
      std::vector<cudf::column_view>{exemplar.child_begin(), exemplar.child_end()}};
  auto deep_copy = std::make_unique<cudf::column>(exemplar_without_null_mask);
  deep_copy->set_null_mask(std::move(null_mask), null_count);
  return deep_copy;
}

std::unique_ptr<cudf::column> generate_list_offsets(cudf::column_view const &list_length,
                                                    rmm::cuda_stream_view stream) {
  CUDF_EXPECTS(list_length.type().id() == cudf::type_id::INT32,
               "Input column does not have type INT32.");

  auto const begin_iter = list_length.template begin<cudf::size_type>();
  auto const end_iter = list_length.template end<cudf::size_type>();

  auto offsets_column = make_numeric_column(data_type{type_id::INT32}, list_length.size() + 1,
                                            mask_state::UNALLOCATED, stream);
  auto offsets_view = offsets_column->mutable_view();
  auto d_offsets = offsets_view.template begin<int32_t>();

  thrust::inclusive_scan(rmm::exec_policy(stream), begin_iter, end_iter, d_offsets + 1);
  CUDF_CUDA_TRY(hipMemsetAsync(d_offsets, 0, sizeof(int32_t), stream));

  return offsets_column;
}

// TODO: Remove these function and use from cudf
namespace {

std::unique_ptr<column> generate_labels(lists_column_view const &input, size_type n_elements,
                                        rmm::cuda_stream_view stream) {
  auto labels = make_numeric_column(data_type(type_to_id<size_type>()), n_elements,
                                    cudf::mask_state::UNALLOCATED, stream);
  auto const labels_begin = labels->mutable_view().template begin<size_type>();
  cudf::detail::label_segments(input.offsets_begin(), input.offsets_end(), labels_begin,
                               labels_begin + n_elements, stream);
  return labels;
}

std::unique_ptr<column> reconstruct_offsets(column_view const &labels, size_type n_lists,
                                            rmm::cuda_stream_view stream)

{
  auto out_offsets = make_numeric_column(data_type{type_to_id<offset_type>()}, n_lists + 1,
                                         mask_state::UNALLOCATED, stream);

  auto const labels_begin = labels.template begin<size_type>();
  auto const offsets_begin = out_offsets->mutable_view().template begin<offset_type>();
  cudf::detail::labels_to_offsets(labels_begin, labels_begin + labels.size(), offsets_begin,
                                  offsets_begin + out_offsets->size(), stream);
  return out_offsets;
}

} // namespace

std::unique_ptr<cudf::column> lists_distinct_by_key(cudf::lists_column_view const &input,
                                                    rmm::cuda_stream_view stream) {
  auto const child = input.get_sliced_child(stream);
  auto const labels = generate_labels(input, child.size(), stream);

  // Use `cudf::duplicate_keep_option::KEEP_LAST` so this will produce the desired behavior when
  // being called in `create_map` in spark-rapids.
  auto distinct_columns =
      cudf::detail::stable_distinct(
          table_view{{labels->view(), child.child(0), child.child(1)}}, // input table
          std::vector<size_type>{0, 1},                                 // key columns
          cudf::duplicate_keep_option::KEEP_LAST, cudf::null_equality::EQUAL,
          cudf::nan_equality::ALL_EQUAL, stream)
          ->release();

  // Assemble a lists column of struct<out_keys, out_vals> for the final output.
  auto out_structs_members = std::vector<std::unique_ptr<cudf::column>>();
  out_structs_members.emplace_back(std::move(distinct_columns[1]));
  out_structs_members.emplace_back(std::move(distinct_columns[2]));
  auto out_structs =
      cudf::make_structs_column(distinct_columns[0]->size(), std::move(out_structs_members), 0, {});

  auto out_offsets = reconstruct_offsets(distinct_columns.front()->view(), input.size(), stream);
  return cudf::make_lists_column(input.size(), std::move(out_offsets), std::move(out_structs),
                                 input.null_count(),
                                 cudf::detail::copy_bitmask(input.parent(), stream), stream);
}

} // namespace cudf::jni
