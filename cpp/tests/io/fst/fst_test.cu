/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <io/fst/lookup_tables.cuh>
#include <io/utilities/hostdevice_vector.hpp>

#include <cudf_test/base_fixture.hpp>
#include <cudf_test/cudf_gtest.hpp>

#include <cudf/scalar/scalar_factories.hpp>
#include <cudf/strings/repeat_strings.hpp>
#include <cudf/types.hpp>

#include <rmm/hip_stream.hpp>
#include <rmm/cuda_stream_view.hpp>
#include <rmm/hip_stream.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>

#include <cstdlib>
#include <vector>

namespace {

//------------------------------------------------------------------------------
// CPU-BASED IMPLEMENTATIONS FOR VERIFICATION
//------------------------------------------------------------------------------
/**
 * @brief CPU-based implementation of a finite-state transducer (FST).
 *
 * @tparam InputItT Forward input iterator type to symbols fed into the FST
 * @tparam StateT Type representing states of the finite-state machine
 * @tparam SymbolGroupLutT Sequence container of symbol groups. Each symbol group is a sequence
 * container to symbols within that group.
 * @tparam TransitionTableT Two-dimensional container type
 * @tparam TransducerTableT Two-dimensional container type
 * @tparam OutputItT Forward output iterator type
 * @tparam IndexOutputItT Forward output iterator type
 * @param[in] begin Forward iterator to the beginning of the symbol sequence
 * @param[in] end Forward iterator to one past the last element of the symbol sequence
 * @param[in] init_state The starting state of the finite-state machine
 * @param[in] symbol_group_lut Sequence container of symbol groups. Each symbol group is a sequence
 * container to symbols within that group. The index of the symbol group containing a symbol being
 * read will be used as symbol_gid of the transition and translation tables.
 * @param[in] transition_table The two-dimensional transition table, i.e.,
 * transition_table[state][symbol_gid] -> new_state
 * @param[in] translation_table The two-dimensional transducer table, i.e.,
 * translation_table[state][symbol_gid] -> range_of_output_symbols
 * @param[out] out_tape A forward output iterator to which the transduced input will be written
 * @param[out] out_index_tape A forward output iterator to which indexes of the symbols that
 * actually caused some output are written to
 * @return A pair of iterators to one past the last element of (1) the transduced output symbol
 * sequence and (2) the indexes of
 */
template <typename InputItT,
          typename StateT,
          typename SymbolGroupLutT,
          typename TransitionTableT,
          typename TransducerTableT,
          typename OutputItT,
          typename IndexOutputItT>
static std::pair<OutputItT, IndexOutputItT> fst_baseline(InputItT begin,
                                                         InputItT end,
                                                         StateT const& init_state,
                                                         SymbolGroupLutT symbol_group_lut,
                                                         TransitionTableT transition_table,
                                                         TransducerTableT translation_table,
                                                         OutputItT out_tape,
                                                         IndexOutputItT out_index_tape)
{
  // Initialize "FSM" with starting state
  StateT state = init_state;

  // To track the symbol offset within the input that caused the FST to output
  std::size_t in_offset = 0;
  for (auto it = begin; it < end; it++) {
    // The symbol currently being read
    auto const& symbol = *it;

    std::size_t symbol_group = 0;
    bool found               = false;

    // Iterate over symbol groups and search for the first symbol group containing the current
    // symbol
    for (auto const& sg : symbol_group_lut) {
      for (auto const& s : sg)
        if (s == symbol) found = true;
      if (found) break;
      symbol_group++;
    }

    // Output the translated symbols to the output tape
    size_t inserted = 0;
    for (auto out : translation_table[state][symbol_group]) {
      // std::cout << in_offset << ": " << out << "\n";
      *out_tape = out;
      ++out_tape;
      inserted++;
    }

    // Output the index of the current symbol, iff it caused some output to be written
    if (inserted > 0) {
      *out_index_tape = in_offset;
      out_index_tape++;
    }

    // Transition the state of the finite-state machine
    state = transition_table[state][symbol_group];

    in_offset++;
  }
  return {out_tape, out_index_tape};
}

//------------------------------------------------------------------------------
// TEST FST SPECIFICATIONS
//------------------------------------------------------------------------------
// FST to check for brackets and braces outside of pairs of quotes
// The state being active while being outside of a string. When encountering an opening bracket
// or curly brace, we push it onto the stack. When encountering a closing bracket or brace, we
// pop it from the stack.
constexpr uint32_t TT_OOS = 0U;

// The state being active while being within a string (e.g., field name or a string value). We do
// not push or pop from the stack while being in this state.
constexpr uint32_t TT_STR = 1U;

// The state being active after encountering an escape symbol (e.g., '\') while being in the TT_STR
// state. constexpr uint32_t TT_ESC = 2U; // cmt to avoid 'unused' warning

// Total number of states
constexpr uint32_t TT_NUM_STATES = 3U;

// Definition of the symbol groups
enum PDA_SG_ID {
  OBC = 0U,          ///< Opening brace SG: {
  OBT,               ///< Opening bracket SG: [
  CBC,               ///< Closing brace SG: }
  CBT,               ///< Closing bracket SG: ]
  QTE,               ///< Quote character SG: "
  ESC,               ///< Escape character SG: '\'
  OTR,               ///< SG implicitly matching all other characters
  NUM_SYMBOL_GROUPS  ///< Total number of symbol groups
};

// Transition table
const std::vector<std::vector<int32_t>> pda_state_tt = {
  /* IN_STATE         {       [       }       ]       "       \    OTHER */
  /* TT_OOS    */ {TT_OOS, TT_OOS, TT_OOS, TT_OOS, TT_STR, TT_OOS, TT_OOS},
  /* TT_STR    */ {TT_STR, TT_STR, TT_STR, TT_STR, TT_OOS, TT_STR, TT_STR},
  /* TT_ESC    */ {TT_STR, TT_STR, TT_STR, TT_STR, TT_STR, TT_STR, TT_STR}};

// Translation table (i.e., for each transition, what are the symbols that we output)
const std::vector<std::vector<std::vector<char>>> pda_out_tt = {
  /* IN_STATE        {      [      }      ]     "  \   OTHER */
  /* TT_OOS    */ {{'{'}, {'['}, {'}'}, {']'}, {'x'}, {'x'}, {'x'}},
  /* TT_STR    */ {{'x'}, {'x'}, {'x'}, {'x'}, {'x'}, {'x'}, {'x'}},
  /* TT_ESC    */ {{'x'}, {'x'}, {'x'}, {'x'}, {'x'}, {'x'}, {'x'}}};

// The i-th string representing all the characters of a symbol group
const std::vector<std::string> pda_sgs = {"{", "[", "}", "]", "\"", "\\"};

// The DFA's starting state
constexpr int32_t start_state = TT_OOS;

}  // namespace

// Base test fixture for tests
struct FstTest : public cudf::test::BaseFixture {
};

TEST_F(FstTest, GroundTruth)
{
  // Type used to represent the atomic symbol type used within the finite-state machine
  using SymbolT = char;

  // Type sufficiently large to index symbols within the input and output (may be unsigned)
  using SymbolOffsetT = uint32_t;

  // Helper class to set up transition table, symbol group lookup table, and translation table
  using DfaFstT = cudf::io::fst::detail::Dfa<char, NUM_SYMBOL_GROUPS, TT_NUM_STATES>;

  // Prepare cuda stream for data transfers & kernels
  rmm::hip_stream stream{};

  // Test input
  std::string input = R"(  {)"
                      R"(category": "reference",)"
                      R"("index:" [4,12,42],)"
                      R"("author": "Nigel Rees",)"
                      R"("title": "Sayings of the Century",)"
                      R"("price": 8.95)"
                      R"(}  )"
                      R"({)"
                      R"("category": "reference",)"
                      R"("index:" [4,{},null,{"a":[]}],)"
                      R"("author": "Nigel Rees",)"
                      R"("title": "Sayings of the Century",)"
                      R"("price": 8.95)"
                      R"(}  {} [] [ ])";

  // Repeat input sample 1024x
  size_t string_size                 = 1 << 10;
  auto d_input_scalar                = cudf::make_string_scalar(input);
  auto& d_string_scalar              = static_cast<cudf::string_scalar&>(*d_input_scalar);
  const cudf::size_type repeat_times = string_size / input.size();
  auto d_input_string                = cudf::strings::repeat_string(d_string_scalar, repeat_times);
  auto& d_input = static_cast<cudf::scalar_type_t<std::string>&>(*d_input_string);
  input         = d_input.to_string(stream);



  // Prepare input & output buffers
  constexpr std::size_t single_item = 1;
  rmm::device_uvector<SymbolT> d_input(input.size(), stream.view());
  hostdevice_vector<SymbolT> output_gpu(input.size(), stream.view());
  hostdevice_vector<SymbolOffsetT> output_gpu_size(single_item, stream.view());
  hostdevice_vector<SymbolOffsetT> out_indexes_gpu(input.size(), stream.view());
  ASSERT_CUDA_SUCCEEDED(hipMemcpyAsync(
    d_input.data(), input.data(), input.size() * sizeof(SymbolT), hipMemcpyHostToDevice, stream.value()));

  // Run algorithm
  DfaFstT parser{pda_sgs, pda_state_tt, pda_out_tt, stream.value()};

  // Allocate device-side temporary storage & run algorithm
  parser.Transduce(d_input.data(),
                   static_cast<SymbolOffsetT>(d_input.size()),
                   output_gpu.device_ptr(),
                   out_indexes_gpu.device_ptr(),
                   output_gpu_size.device_ptr(),
                   start_state,
                   stream.value());

  // Async copy results from device to host
  output_gpu.device_to_host(stream.view());
  out_indexes_gpu.device_to_host(stream.view());
  output_gpu_size.device_to_host(stream.view());

  // Prepare CPU-side results for verification
  std::string output_cpu{};
  std::vector<SymbolOffsetT> out_index_cpu{};
  output_cpu.reserve(input.size());
  out_index_cpu.reserve(input.size());

  // Run CPU-side algorithm
  fst_baseline(std::begin(input),
               std::end(input),
               start_state,
               pda_sgs,
               pda_state_tt,
               pda_out_tt,
               std::back_inserter(output_cpu),
               std::back_inserter(out_index_cpu));

  // Make sure results have been copied back to host
  stream.synchronize();

  // Verify results
  ASSERT_EQ(output_gpu_size[0], output_cpu.size());
  ASSERT_EQ(out_indexes_gpu.size(), out_index_cpu.size());
  for (std::size_t i = 0; i < output_cpu.size(); i++) {
    ASSERT_EQ(output_gpu[i], output_cpu[i]) << "Mismatch at index #" << i;
  }
  for (std::size_t i = 0; i < out_indexes_gpu.size(); i++) {
    ASSERT_EQ(out_indexes_gpu[i], out_index_cpu[i]) << "Mismatch at index #" << i;
  }
}

CUDF_TEST_PROGRAM_MAIN()
